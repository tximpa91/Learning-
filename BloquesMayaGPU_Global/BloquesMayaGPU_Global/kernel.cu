#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <stdlib.h>
#include <stdio.h>
#include <Windows.h>
#include <time.h>



__device__ void comprobar(int *matriz, int fil, int col, int columna) //metodo para recorrer toda la matriz en busca de posiciones vacias
{																	// para desplazarlas arriba		

	for (int k = 0; k < columna; k++)
	{
		if (col> 0)
		{
			if (matriz[col*columna + fil] == 0 && !matriz[(col - 1)*columna + fil] == 0) // Si hay una posicon vacia encima de otra no lo hace
			{
				int aux = matriz[col*columna + fil];
				matriz[col*columna + fil] = matriz[(col - 1)*columna + fil];
				matriz[(col - 1)*columna + fil] = 0;
			}
		}

		__syncthreads();
	}

}


__device__ void DesplazamientoDerecha(int *matriz, int  fila, int columna, int fil, int col2)
{

	int aux;
	for (int p = 0; p <= fila; p++){ // Para desplazar todas las posiciones de la columna
		if (matriz[(fila)*columna + col2] == 0) //Comprobar la ultima fila
		{
			for (int k = 0; k < columna; k++)
			{
				if (col2 > 0) //En orden
				{
					if (matriz[fil *columna + col2] == 0) //swap
					{
						int aux = matriz[fil *columna + (col2 - 1)];
						matriz[fil *columna + (col2 - 1)] = 0;
						matriz[fil *columna + col2] = aux;
					}

				}

				__syncthreads();
			}

		}
		__syncthreads(); //Despues de cada interacion, se sincronizan para esperar que los hilos hagan sus tareas.
	}
}




int *crearMatriz(int filas, int columna)
{
	int *matriz = (int *)malloc(filas * columna * sizeof(int));
	int contador = 0;
	for (int i = 0; i < filas; i++)
	{
		for (int j = 0; j < columna; j++)
		{
			int numero = rand() % 8 + 1;
			if (numero == 8 && contador < 2) //Creamos 2 bombas
			{
				contador++;
				matriz[i*columna + j] = numero;
			}
			else
			{
				if (numero != 8)
				{
					matriz[i*columna + j] = numero;
				}
				else
				{
					matriz[i*columna + j] = rand() % 7 + 1;
				}

			}
		}
	}

	return matriz;

}

void color(int opcion)
{
	HANDLE hConsole;
	hConsole = GetStdHandle(STD_OUTPUT_HANDLE);


	switch (opcion)
	{
	case 0: {	SetConsoleTextAttribute(hConsole, 0); printf("%d", opcion);
		SetConsoleTextAttribute(hConsole, 7); printf("|");
		break;
	}
	case 1: {	SetConsoleTextAttribute(hConsole, 9); printf("%d", opcion);
		SetConsoleTextAttribute(hConsole, 7); printf("|");
		break;

	}

	case 2: {	 SetConsoleTextAttribute(hConsole, 4); printf("%d", opcion);
		SetConsoleTextAttribute(hConsole, 7); printf("|");
		break;
	}

	case 3: { SetConsoleTextAttribute(hConsole, 14); printf("%d", opcion);
		SetConsoleTextAttribute(hConsole, 7); printf("|");
		break; }

	case 4: {	SetConsoleTextAttribute(hConsole, 2);  printf("%d", opcion);
		SetConsoleTextAttribute(hConsole, 7); printf("|");
		break;
	}

	case 5:{	SetConsoleTextAttribute(hConsole, 13);  printf("%d", opcion);
		SetConsoleTextAttribute(hConsole, 7); printf("|");
		break;
	}

	case 6: {	SetConsoleTextAttribute(hConsole, 6);  printf("%d", opcion);
		SetConsoleTextAttribute(hConsole, 7); printf("|");
		break;
	}

	case 7:	{	SetConsoleTextAttribute(hConsole, 1);  printf("%d", opcion);
		SetConsoleTextAttribute(hConsole, 7); printf("|");
		break;
	}

	case 8: {	SetConsoleTextAttribute(hConsole, 7);  printf("%d", opcion);
		SetConsoleTextAttribute(hConsole, 7); printf("|");
		break;
	}
	case 9: { SetConsoleTextAttribute(hConsole, 0);  printf("%d", opcion);
		SetConsoleTextAttribute(hConsole, 7); printf("|");
		break; }

	default: { printf("ERROR\n"); break; }

	}



}

void imprimir(int *matriz, int filas, int columna)
{

	HANDLE hConsole;
	hConsole = GetStdHandle(STD_OUTPUT_HANDLE);
	//system("cls");
	for (int i = 0; i < filas; i++)
	{
		for (int j = 0; j <columna; j++)
			color(matriz[i*columna + j]);

		printf("\n");


	}

	SetConsoleTextAttribute(hConsole, 7);

}

boolean  posiValida(int fila, int columna, int i, int j) //Comprueba que la posicion metida por teclado sea correcta
{
	if ((i >= 0 && j >= 0) && (i <= fila - 1 && j <= columna - 1))
		return true;
	else
		return false;
}
__device__ void juegarriba(int *matriz, int tid1, int columna, int i, int j, int  bloque)
{

	switch (tid1) //Dos hilos que comprueben la posicion de arriba y la de encima de esta. 
	{
	case 1:	 if (bloque == matriz[(i - tid1)*columna + j])
	{
		matriz[i*columna + j] = 0;
		matriz[(i - tid1)*columna + j] = 0;
	}
			 break;
			 // Si la posicion de arriba es igual, se comprueba la de m�s arriba (para borrar 3)
	case 2:  if (bloque == matriz[(i - tid1)*columna + j] && bloque == matriz[(i - tid1 + 1)*columna + j])
	{
		matriz[i*columna + j] = 0;
		matriz[(i - tid1)*columna + j] = 0;
	}
			 break;
	}


}




__device__ void juegabajo(int *matriz, int tid1, int columna, int i, int j, int  bloque)
{

	switch (tid1)
	{
	case 1:	 if (bloque == matriz[(i + tid1)*columna + j])
	{
		matriz[i*columna + j] = 0;
		matriz[(i + tid1)*columna + j] = 0;
	}
			 break;
			 // Comprueba si la posicion de abajo es igual y si es comprueba la de arriba de esta posicion
	case 2:  if (bloque == matriz[(i + tid1)*columna + j] && matriz[(i + tid1 - 1)*columna + j] == bloque)
	{
		matriz[i*columna + j] = 0;
		matriz[(i + tid1)*columna + j] = 0;
	}
			 break;
	}
}

__device__ void juegaderecha(int *matriz, int tid, int columna, int i, int j, int  bloque)
{
	switch (tid)
	{
	case 1:	 if (bloque == matriz[(i)*columna + (j + tid)])
	{
		matriz[i*columna + j] = 0;
		matriz[i*columna + (j + tid)] = 0;
	}
			 break;

			 //Comprueba la posicion de su derecha y si es 0 comprueba a su vez la de la derecha
	case 2:  if (bloque == matriz[i*columna + (j + tid)] && bloque == matriz[i*columna + (j + tid - 1)])
	{
		matriz[i*columna + j] = 0;
		matriz[i*columna + (j + tid)] = 0;
	}
			 break;
	}


}

__device__ void juegaizquierda(int *matriz, int tid, int columna, int i, int j, int  bloque)
{
	switch (tid)
	{
	case 1:	 if (bloque == matriz[(i)*columna + (j - tid)])
	{
		matriz[i*columna + j] = 0;
		matriz[i*columna + (j - tid)] = 0;
	}
			 break;
			 //Comprueba la posicion de su izquierda y si es 0 comprueba a su vez la de la izquierda
	case 2:  if (bloque == matriz[i*columna + (j - tid)] && bloque == matriz[i*columna + (j - tid + 1)])
	{
		matriz[i*columna + j] = 0;
		matriz[i*columna + (j - tid)] = 0;
	}
			 break;
	}


}


__device__ void bombasabajo(int *matriz, int columna, int i, int j)
{
	matriz[i*columna + j] = 0;// Su misma posicion
	matriz[(i + 1)*columna + j] = 0; // Eliminar abajo
}

__device__ void bombasarriba(int *matriz, int columna, int i, int j)
{
	matriz[i*columna + j] = 0;// Su misma posicion
	matriz[(i - 1)*columna + j] = 0; //Eliminar arriba

}

__device__ void bombasderecha(int *matriz, int columna, int i, int j)
{
	matriz[i*columna + j] = 0;// Su misma posicion
	matriz[i*columna + j + 1] = 0; // Elimnar derecha


}

__device__ void bombasizquierda(int *matriz, int columna, int i, int j)
{
	matriz[i*columna + j] = 0;// Su misma posicion
	matriz[i*columna + j - 1] = 0; //Elimanar izquierda


}

__device__ void bombasDderechaA(int *matriz, int columna, int i, int j)
{
	matriz[i*columna + j] = 0;// Su misma posicion
	matriz[(i - 1)*columna + j + 1] = 0; //Diagonal superior Derecha


}

__device__ void bombasIzquierdaA(int *matriz, int columna, int i, int j)
{
	matriz[i*columna + j] = 0;// Su misma posicion
	matriz[(i - 1)*columna + j - 1] = 0; //Diagonal  superior Izquierda


}

__device__ void bombasDerechaB(int *matriz, int columna, int i, int j)
{
	matriz[i*columna + j] = 0;// Su misma posicion
	matriz[(i + 1)*columna + j + 1] = 0; //Diagonal  inferior Derecha

}

__device__ void bombasIzquierdaB(int *matriz, int columna, int i, int j)
{
	matriz[i*columna + j] = 0;// Su misma posicion
	matriz[(i - 1)*columna + j - 1] = 0;//Diagonal inferior Izquierda


}



__global__ void KERNEL(int *matriz, int fila, int columna, int i, int j)
{

	int fil = threadIdx.y;
	int col = threadIdx.x;
	int bloque = matriz[i*columna + j]; //posicion que hemos seleccionado por pantalla
	printf("");
	if (bloque == 8) //Se comprueba si se ha pulsado una bomba
	{
		if (i == 0 && j == 0) // esquina superior izquierda
		{
			bombasabajo(matriz, columna, i, j);
			bombasderecha(matriz, columna, i, j);
			bombasDerechaB(matriz, columna, i, j);

		}
		else
		{
			if (i == fila - 1 && j == columna - 1) // esquina inferior derecha
			{
				bombasarriba(matriz, columna, i, j);
				bombasizquierda(matriz, columna, i, j);
				bombasIzquierdaA(matriz, columna, i, j);
			}
			else{

				if (i == fila - 1 && j == 0) // esquina inferior izquierda
				{
					bombasarriba(matriz, columna, i, j);
					bombasderecha(matriz, columna, i, j);
					bombasDderechaA(matriz, columna, i, j);

				}

				else
				{

					if (i == 0 && j == columna - 1) // esquina superior derecha
					{
						bombasabajo(matriz, columna, i, j);
						bombasizquierda(matriz, columna, i, j);
						bombasIzquierdaB(matriz, columna, i, j);

					}
					else
					{
						if (i == 0 && j < columna - 1) // primera fila
						{
							bombasabajo(matriz, columna, i, j);
							bombasizquierda(matriz, columna, i, j);
							bombasderecha(matriz, columna, i, j);
							bombasDerechaB(matriz, columna, i, j);
							bombasIzquierdaB(matriz, columna, i, j);

						}
						else
						{

							if (i < fila - 1 && j == 0) // primera columna
							{
								bombasarriba(matriz, columna, i, j);
								bombasderecha(matriz, columna, i, j);
								bombasabajo(matriz, columna, i, j);
								bombasDderechaA(matriz, columna, i, j);
								bombasDerechaB(matriz, columna, i, j);



							}

							else
							{

								if (i < fila - 1 && j == columna - 1) //Ultima columna
								{
									bombasarriba(matriz, columna, i, j);
									bombasabajo(matriz, columna, i, j);
									bombasizquierda(matriz, columna, i, j);
									bombasIzquierdaA(matriz, columna, i, j);
									bombasIzquierdaB(matriz, columna, i, j);


								}
								else
								{
									if (i == fila - 1 && j != 0 && j != columna - 1) // ultima fila
									{
										bombasarriba(matriz, columna, i, j);
										bombasderecha(matriz, columna, i, j);
										bombasizquierda(matriz, columna, i, j);
										bombasDderechaA(matriz, columna, i, j);
										bombasIzquierdaA(matriz, columna, i, j);


									}
									else
									{

										if (i < fila - 1 && j < columna - 1)
										{
											bombasabajo(matriz, columna, i, j);
											bombasarriba(matriz, columna, i, j);
											bombasderecha(matriz, columna, i, j);
											bombasizquierda(matriz, columna, i, j);
											bombasDderechaA(matriz, columna, i, j);
											bombasDerechaB(matriz, columna, i, j);
											bombasIzquierdaA(matriz, columna, i, j);
											bombasIzquierdaB(matriz, columna, i, j);
										}
									}
								}
							}
						}
					}
				}
			}
		}
	}
	else
	{
		if (i == 0 && j == 0) // esquina superior izquierda
		{
			juegabajo(matriz, fil, columna, i, j, bloque);
			juegaderecha(matriz, col, columna, i, j, bloque);
		}
		else
		{
			if (i == fila - 1 && j == columna - 1) // esquina inferior derecha
			{
				juegarriba(matriz, fil, columna, i, j, bloque);
				juegaizquierda(matriz, col, columna, i, j, bloque);
			}
			else{

				if (i == fila - 1 && j == 0) // esquina inferior izquierda
				{
					juegarriba(matriz, fil, columna, i, j, bloque);
					juegaderecha(matriz, col, columna, i, j, bloque);

				}

				else
				{

					if (i == 0 && j == columna - 1) // esquina superior derecha
					{
						juegaizquierda(matriz, fil, columna, i, j, bloque);
						juegabajo(matriz, fil, columna, i, j, bloque);
					}
					else
					{
						if (i == 0 && j < columna - 1) // primera fila
						{
							juegaizquierda(matriz, fil, columna, i, j, bloque);
							juegaderecha(matriz, fil, columna, i, j, bloque);
							juegabajo(matriz, fil, columna, i, j, bloque);
						}
						else
						{

							if (i < fila - 1 && j == 0) // primera columna
							{
								juegarriba(matriz, fil, columna, i, j, bloque);
								juegabajo(matriz, fil, columna, i, j, bloque);
								juegaderecha(matriz, col, columna, i, j, bloque);

							}

							else
							{

								if (i < fila - 1 && j == columna - 1) //Ultima columna
								{
									juegarriba(matriz, fil, columna, i, j, bloque);
									juegabajo(matriz, fil, columna, i, j, bloque);
									juegaizquierda(matriz, col, columna, i, j, bloque);

								}
								else
								{
									if (i == fila - 1 && j != 0 && j != columna - 1) // ultima fila
									{
										juegarriba(matriz, fil, columna, i, j, bloque);
										juegaderecha(matriz, col, columna, i, j, bloque);
										juegaizquierda(matriz, fil, columna, i, j, bloque);

									}
									else
									{

										if (i < fila - 1 && j < columna - 1)
										{
											juegarriba(matriz, fil, columna, i, j, bloque);
											juegabajo(matriz, fil, columna, i, j, bloque);
											juegaderecha(matriz, col, columna, i, j, bloque);
											juegaizquierda(matriz, fil, columna, i, j, bloque);
										}
									}
								}
							}
						}
					}
				}
			}
		}
	}
	__syncthreads();
	comprobar(matriz, fil, col, columna); //Se recorre la matriz en busca de 0 para desplazarlos para arriba


	__syncthreads();
	DesplazamientoDerecha(matriz, fila - 1, columna, fil, col);



}









void test(){
	hipDeviceProp_t prop;
	int count;
	(hipGetDeviceCount(&count));
	for (int i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf(" --- General Information for device %d ---\n", i);
		printf("Name: %s\n", prop.name);
		printf("Compute capability: %d.%d\n", prop.major, prop.minor);
		printf("Clock rate: %d\n", prop.clockRate);
		printf("Device copy overlap: ");
		if (prop.deviceOverlap)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("Kernel execition timeout : ");
		if (prop.kernelExecTimeoutEnabled)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf(" --- Memory Information for device %d ---\n", i);
		printf("Total global mem: %ld\n", prop.totalGlobalMem);
		printf("Total constant Mem: %ld\n", prop.totalConstMem);
		printf("Max mem pitch: %ld\n", prop.memPitch);
		printf("Texture Alignment: %ld\n", prop.textureAlignment);
		printf(" --- MP Information for device %d ---\n", i);
		printf("Multiprocessor count: %d\n",
			prop.multiProcessorCount);
		printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp: %d\n", prop.regsPerBlock);
		printf("Threads in warp: %d\n", prop.warpSize);
		printf("Max threads per block: %d\n",
			prop.maxThreadsPerBlock);
		printf("Max thread dimensions: (%d, %d, %d)\n",
			prop.maxThreadsDim[0], prop.maxThreadsDim[1],
			prop.maxThreadsDim[2]);
		printf("Max grid dimensions: (%d, %d, %d)\n",
			prop.maxGridSize[0], prop.maxGridSize[1],
			prop.maxGridSize[2]);
		printf("\n");
	}
}


int main()
{
	int opcion = -1, i = 0, j = 0;
	int *matriz, *matriz_CUDA, *matrizC;
	int filas = -1;
	int columna = -1;
	char modo = ' ';
	time_t t;
	srand((unsigned)time(&t));
	boolean valido = false;



	printf("Bienvenidos a Antique Blocks \n");

	printf(" -------------- Elija una Modalidad de Juego ----------------\n ");

	while (modo != 'a' && modo != 'm')
	{

		printf(" < a > Juego Automatico \n");
		printf(" < m > Juego Manual \n");
		modo = getchar();

	}

	while (filas < 0 && columna < 0)
	{
		system("cls");
		printf(" Inutroduzca la cantidad de filas: \n");
		scanf("%d", &filas);
		printf(" Introduzca la cantidad de columnas: \n");
		scanf("%d", &columna);
		matriz = crearMatriz(filas, columna);
		system("cls");


	}

	if (modo == 'm') {
		int size = filas * columna * sizeof(int);

		dim3 DimGrid(1, 1);
		dim3 DimBlock(filas, columna, 1);
		hipMalloc((void**)&matriz_CUDA, size);
		hipMemcpy(matriz_CUDA, matriz, size, hipMemcpyHostToDevice);//Mandamos la matriz a GPU

		while (true)
		{

			valido = false;
			while (!valido)
			{
				imprimir(matriz, filas, columna);
				printf("Introduzca la casilla a jugar \n");
				printf("Fila: ");
				scanf("%d", &i);
				printf("\n");
				printf("Columna ");
				scanf("%d", &j);
				printf("\n");
				i--;
				j--;
				valido = posiValida(filas, columna, i, j);
				if (valido == false)
				{
					printf("Posicion invalida \n");
					system("pause");
					system("cls");

				}

			}





			KERNEL << <DimGrid, DimBlock >> >(matriz_CUDA, filas, columna, i, j);


			hipMemcpy(matriz, matriz_CUDA, size, hipMemcpyDeviceToHost); //metemos la matriz en CPU


			system("cls");
		}
		hipFree(matriz_CUDA);
		system("pause");
	}
	else
	{

		int size = filas * columna * sizeof(int);

		dim3 DimGrid(1, 1);
		dim3 DimBlock(filas, columna, 1);
		hipMalloc((void**)&matriz_CUDA, size);
		hipMemcpy(matriz_CUDA, matriz, size, hipMemcpyHostToDevice);//Mandamos la matriz a GPU
		int aux, aux2;
		for (int i = 0; i < filas; i++)
		{
			for (int j = 0; j < columna; j++)
			{
				aux = i + 1;
				aux2 = j + 1;
				printf("Posicion a Jugar  fila: %d  columna: %d \n", aux, aux2);
				KERNEL << <DimGrid, DimBlock >> >(matriz_CUDA, filas, columna, i, j);


				hipMemcpy(matriz, matriz_CUDA, size, hipMemcpyDeviceToHost); //metemos la matriz en CPU

				imprimir(matriz, filas, columna);
				system("pause");
				system("cls");


			}


		}

	}


	return 0;



}